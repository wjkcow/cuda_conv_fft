#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cuda_fft_lib.h"
#include "helper.h"


inline __device__ void mulAndScale(fComplex &a, fComplex &b, fComplex &t, float c)
{
    t.x += c *(a.x * b.x - a.y * b.y);
    t.y += c *(a.y * b.x + a.x * b.y);
}

// z = x .* y / c channel added together
__global__ void cuda_multi_add(fComplex * x_base, fComplex * y, fComplex * z_base,
	 int xy, int ch){

	int img_stack      =  blockIdx.x;  // 1 - 5000

	fComplex *x = x_base + img_stack * xy * ch;
	fComplex *z = z_base + img_stack * xy * ch;
	for(int c = 0; c < ch ; c++){
    	for(int i = 0; i < xy ; i ++){
    		mulAndScale(x[c * xy + i], y[c * xy + i], z[xy + i], 1.0/xy);
    	}
	}
}

extern "C" void cuda_init(){
	hipSetDevice(0);
	DEVICE_RESET
}

extern "C" fComplex* batch_fft2(float * gpuIn, int x, int y, int batch){
	hipfftHandle fftPlanFwd;
	fComplex *gpuOut;
	int dim[2];
	dim[0] = x;
	dim[1] = y;
	int fftSize = x * y * batch;
	if (hipfftPlanMany(&fftPlanFwd, 2, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_R2C, batch) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;	
	}

	checkCudaErrors(hipMalloc((void **)&gpuOut,  fftSize * sizeof(fComplex)));
	checkCudaErrors(hipMemset(gpuOut, 0, fftSize * sizeof(fComplex)));
	checkCudaErrors(hipfftExecR2C(fftPlanFwd, (hipfftReal *)gpuIn, (hipfftComplex *)gpuOut));
	checkCudaErrors(hipDeviceSynchronize());

	return gpuOut;
}
extern "C" bool multi_sum_norm(fComplex * img, fComplex * filter, fComplex * result,
	int xy, int ch, int img_batch){

	dim3 grid(img_batch, 1);
	dim3 block(1);
	cuda_multi_add<<<grid,block,0,0>>>(img, filter, result, xy, ch);
	checkCudaErrors(hipDeviceSynchronize());
	return true;
}
extern "C" bool batch_ifft2(fComplex * gpuIn, float * gpuOut, int x, int y, int batch){
	hipfftHandle fftPlanInv;
	int dim[2];
	dim[0] = x;
	dim[1] = y;
//	int fftSize = x * y * batch;
	if (hipfftPlanMany(&fftPlanInv, 2, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2R,batch) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;	
	}
	checkCudaErrors(hipfftExecC2R(fftPlanInv, (hipfftComplex *)gpuIn,(hipfftReal *)gpuOut));
    checkCudaErrors(hipDeviceSynchronize());

	return true;
}










