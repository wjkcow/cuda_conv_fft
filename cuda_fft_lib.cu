#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cuda_fft_lib.h"
#include "helper.h"


inline __device__ void mulAndScale(fComplex &a, fComplex &b, fComplex &t, float c)
{
    t.x += c *(a.x * b.x - a.y * b.y);
    t.y += c *(a.y * b.x + a.x * b.y);
}

// z = x .* y / c channel added together
__global__ void cuda_multi_add(fComplex * x_base, fComplex * y, fComplex * z_base,
	 int xy, int ch, float ratio){

	int img_stack      =  blockIdx.x;  // 1 - 5000

	fComplex *x = x_base + img_stack * xy * ch;
	fComplex *z = z_base + img_stack * xy;
	for(int c = 0; c < ch ; c++){
    	for(int i = 0; i < xy ; i ++){
    		mulAndScale(x[c * xy + i], y[c * xy + i], z[i], 1.0/ratio);
    	}
	}
}

extern "C" void cuda_init(int n){
	hipSetDevice(n);
	DEVICE_RESET
}

extern "C" fComplex* batch_fft2(float * gpuIn, int x, int y, int batch){
	hipfftHandle fftPlanFwd;
	fComplex *gpuOut;
	int dim[2];
	dim[0] = x;
	dim[1] = y;
	int fftSize = x * (y / 2 +1)* batch;
	if (hipfftPlanMany(&fftPlanFwd, 2, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_R2C, batch) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;	
	}

	checkCudaErrors(hipMalloc((void **)&gpuOut,  fftSize * sizeof(fComplex)));
	checkCudaErrors(hipMemset(gpuOut, 0, fftSize * sizeof(fComplex)));
	checkCudaErrors(hipfftExecR2C(fftPlanFwd, (hipfftReal *)gpuIn, (hipfftComplex *)gpuOut));
	checkCudaErrors(hipDeviceSynchronize());
	hipfftDestroy(fftPlanFwd);
	return gpuOut;
}
extern "C" bool multi_sum_norm(fComplex * img, fComplex * filter, fComplex * result,
	int xy, int ch, int img_batch, float ratio){

	dim3 grid(img_batch, 1);
	dim3 block(1);
	cuda_multi_add<<<grid,block,0,0>>>(img, filter, result, xy, ch, ratio);
	checkCudaErrors(hipDeviceSynchronize());
	return true;
}
extern "C" bool batch_ifft2(fComplex * gpuIn, float * gpuOut, int x, int y, int batch){
	hipfftHandle fftPlanInv;
	int dim[2];
	dim[0] = x;
	dim[1] = y;
//	int fftSize = x * y * batch;
	if (hipfftPlanMany(&fftPlanInv, 2, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2R,batch) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;	
	}
	checkCudaErrors(hipfftExecC2R(fftPlanInv, (hipfftComplex *)gpuIn,(hipfftReal *)gpuOut));
    checkCudaErrors(hipDeviceSynchronize());
	hipfftDestroy(fftPlanInv);
	return true;
}



extern "C" bool conv_cufft(float * cpuImg, float * cpuFilter, float *ans,
	int x, int y, int ch, int batch, int f_batch){

	float * gpuImg, * gpuFilter, * gpuAns;
	fComplex *gpuImg_C, *gpuFilter_C, *gpuProd;
	size_t img_size = (size_t) x * y * ch * batch;
	size_t filter_size = (size_t) x * y * ch * f_batch;
	size_t ans_size = (size_t) x * y * batch;
	size_t conv_size = (size_t) x * (y/2 + 1) * batch;
//
//	fComplex *cpuProd =(fComplex *) malloc(ans_size * sizeof(fComplex));
//
//	cuda_init();
	checkCudaErrors(hipMalloc((void **)&gpuProd,  conv_size * sizeof(fComplex)));
	checkCudaErrors(hipMalloc((void **)&gpuImg,  img_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&gpuFilter,  filter_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&gpuAns,  ans_size * sizeof(float)));

	checkCudaErrors(hipMemcpy(gpuImg, cpuImg, img_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpuFilter, cpuFilter, filter_size * sizeof(float), hipMemcpyHostToDevice));

	gpuImg_C = batch_fft2(gpuImg, x, y, ch * batch );
	gpuFilter_C = batch_fft2(gpuFilter, x, y, ch * f_batch );
	float * my_cpu_ans = ans;
	for(int b = 0; b < f_batch; ++ b){
		fprintf(stderr, "batch %d\n", b);
		checkCudaErrors(hipMemset(gpuProd, 0, conv_size * sizeof(fComplex)));
		fComplex *myFilter = gpuFilter_C + b * x * (y / 2 + 1) * ch;
		multi_sum_norm(gpuImg_C, myFilter, gpuProd, x*(y / 2 + 1) , ch, batch, (float)x*y);
//		checkCudaErrors(hipMemcpy(cpuProd, gpuProd, ans_size * sizeof(float), hipMemcpyDeviceToHost));
//
	//	for(int j = 0; j < 18; ++j){
		//	fprintf(stderr, "%d %f, %f \n",j,cpuProd[j].x , cpuProd[j].y );
		//}

		batch_ifft2(gpuProd, gpuAns , x, y, batch);
		my_cpu_ans = my_cpu_ans + x * y * batch;
		fprintf(stderr,"copying to %p",my_cpu_ans);
		checkCudaErrors(hipMemcpy(my_cpu_ans, gpuAns, x * y * batch* sizeof(float), hipMemcpyDeviceToHost));
		fprintf(stderr, "copy end");
	}

	checkCudaErrors(hipFree(gpuProd));
	checkCudaErrors(hipFree(gpuImg));
	checkCudaErrors(hipFree(gpuFilter));
	checkCudaErrors(hipFree(gpuAns));
	checkCudaErrors(hipFree(gpuImg_C));
	checkCudaErrors(hipFree(gpuFilter_C));
	return true;
}






