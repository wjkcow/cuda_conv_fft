#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cuda_fft_lib.h"
#include "helper.h"


inline __device__ void mulAndScale(fComplex &a, fComplex &b, fComplex &t, float c)
{
    t.x += c *(a.x * b.x - a.y * b.y);
    t.y += c *(a.y * b.x + a.x * b.y);
}

// z = x .* y / c channel added together
__global__ void cuda_multi_add(fComplex * x_base, fComplex * y, fComplex * z_base,
	 int xy, int ch){

	int img_stack      =  blockIdx.x;  // 1 - 5000

	fComplex *x = x_base + img_stack * xy * ch;
	fComplex *z = z_base + img_stack * xy;
	for(int c = 0; c < ch ; c++){
    	for(int i = 0; i < xy ; i ++){
    		mulAndScale(x[c * xy + i], y[c * xy + i], z[i], 1.0/xy/xy);
    	}
	}
}

extern "C" void cuda_init(){
	hipSetDevice(0);
	DEVICE_RESET
}

extern "C" fComplex* batch_fft2(float * gpuIn, int x, int y, int batch){
	hipfftHandle fftPlanFwd;
	fComplex *gpuOut;
	int dim[2];
	dim[0] = x;
	dim[1] = y;
	int fftSize = x * y * batch;
	if (hipfftPlanMany(&fftPlanFwd, 2, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_R2C, batch) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;	
	}

	checkCudaErrors(hipMalloc((void **)&gpuOut,  fftSize * sizeof(fComplex)));
	checkCudaErrors(hipMemset(gpuOut, 0, fftSize * sizeof(fComplex)));
	checkCudaErrors(hipfftExecR2C(fftPlanFwd, (hipfftReal *)gpuIn, (hipfftComplex *)gpuOut));
	checkCudaErrors(hipDeviceSynchronize());

	return gpuOut;
}
extern "C" bool multi_sum_norm(fComplex * img, fComplex * filter, fComplex * result,
	int xy, int ch, int img_batch){

	dim3 grid(img_batch, 1);
	dim3 block(1);
	cuda_multi_add<<<grid,block,0,0>>>(img, filter, result, xy, ch);
	checkCudaErrors(hipDeviceSynchronize());
	return true;
}
extern "C" bool batch_ifft2(fComplex * gpuIn, float * gpuOut, int x, int y, int batch){
	hipfftHandle fftPlanInv;
	int dim[2];
	dim[0] = x;
	dim[1] = y;
//	int fftSize = x * y * batch;
	if (hipfftPlanMany(&fftPlanInv, 2, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2R,batch) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;	
	}
	checkCudaErrors(hipfftExecC2R(fftPlanInv, (hipfftComplex *)gpuIn,(hipfftReal *)gpuOut));
    checkCudaErrors(hipDeviceSynchronize());

	return true;
}



extern "C" bool conv_cufft(float * cpuImg, float * cpuFilter, float *ans,
	int x, int y, int ch, int batch, int f_batch){

	float * gpuImg, * gpuFilter, * gpuAns, *gpuProd;
	fComplex *gpuImg_C, *gpuFilter_C;
	size_t img_size = (size_t) x * y * ch * batch;
	size_t filter_size = (size_t) x * y * ch * f_batch;
	size_t ans_size = (size_t) x * y * batch;

	cuda_init();
	checkCudaErrors(hipMalloc((void **)&gpuProd,  ans_size * sizeof(fComplex)));
	checkCudaErrors(hipMalloc((void **)&gpuImg,  img_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&gpuFilter,  filter_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&gpuAns,  ans_size * sizeof(float)));

	checkCudaErrors(hipMemcpy(gpuImg, cpuImg, img_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpuFilter, fiter, filter_size * sizeof(float), hipMemcpyHostToDevice));

	gpuImg_C = batch_fft2(gpuImg, x, y, ch * batch );
	gpuFilter_C = batch_fft2(gpuFilter, x, y, ch * f_batch );

	for(int b = 0; b < f_batch; ++ b){
		fComplex *myFilter = gpuFilter_C + b * x * y * ch;
		multi_sum_norm(gpuImg_C, gpuFilter_C, gpuProd, x*y , ch, batch);
		batch_ifft2(gpuProd, gpuAns , x, y, batch);
		float *my_cpu_ans = ans + b * x * y * batch;
		checkCudaErrors(hipMemcpy(gpuR1, gpuAns, x * y * batch* sizeof(float), hipMemcpyDeviceToHost));
	}

	checkCudaErrors(hipFree(gpuProd));
	checkCudaErrors(hipFree(gpuImg));
	checkCudaErrors(hipFree(gpuFilter));
	checkCudaErrors(hipFree(gpuAns));
	checkCudaErrors(hipFree(gpuImg_C));
	checkCudaErrors(hipFree(gpuFilter_C));
	return true;
}






